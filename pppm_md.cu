/*
	This code is using Molecular dynamics to study the evolution of the electron bunch form the photo cathod in Ultrafast electron diffraction experiment.

	origian Fortran code version: MD_1121.f90

*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <omp.h>
//#include <fftw3.h>
#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

//simulation setup
#define N 5000					//number of particles
#define Ntime 1 
#define newR 2.0E6		//size of the simulation box ~ 100 um
#define cutoff 200.0	//lower limit of the initial distance between electrons ~ 100 nm

//PPPM setup
#define bn 100					//number of boxes per direction
#define boxcap 5		//temporary cap for particles in one box; update to class later
#define hx 2.0E4			//[real space] cell size newR/bn
#define hx3 8.0E12		//hx^3 as the cell volume
#define bn3 1000000			// total grid/mesh point number

//parameters
#define m 5.4858e-4		//elelctron mass
#define vc 5.85e3			//speed of light
#define PI 3.141592653589793	//  \pi

double inline getrand(){ return (double)rand()/(double)RAND_MAX; }

//***************
__global__ void theKernel(double * r_d){
  //This is array flattening, (Array Width * Y Index + X Index)
  r_d[(gridDim.x * blockDim.x) * (blockIdx.y * blockDim.y + threadIdx.y) + (blockIdx.x * blockDim.x + threadIdx.x)] += 5;
}

void printGrid(double a[N][3]){
  for (int i = 0; i < 5; i++) {
    for (int j = 0; j < 3; j++) {
      printf("%11.3f ",a[i][j]);
    }
    printf("\n");
  }
}


//****************
double getPE(double r[N][3]){
	int i,j,k;
	double rel,vij;
	double PE_c = 0.0;
	for(j=0; j<(N-1); j++) {
		for(i=j+1; i<N; i++) {
			vij = 0.0;
			for(k=0; k<3; k++) {
				rel = r[i][k] - r[j][k];
				vij += pow(rel,2.0); //vij = r^2 for now
			}
			PE_c += pow(vij,-0.5); // r^(-1)
		}
	}
	return PE_c;
}

double getKE(double v[N][3]){
	int i,j;
	double KE_c = 0.0; //sum(v**2)
	for (i=0; i<N; i++){
		for (j=0; j<3; j++) {
			KE_c += pow(v[i][j],2.0);
		}
	}
	KE_c = 0.5*m*KE_c;
	return KE_c;
}

int inline pbc_box(int point){
	int pb;
	if (point < 0 ) {
		pb = point + bn;
	}
	else if (point > (bn-1)) {
		pb = point - bn;
	}
	else {
		pb = point;
	}
	return pb;
}

void charge_assign(int b[3], double basis[3],double rho[bn][bn][bn],int id,double w[N][2][2][2]){
	int dx,dy,dz,pbx,pby,pbz;
	double lx,ly,lz,weight;
	for (dx=0; dx<2; dx++){
		pbx = pbc_box(b[0]+dx);
	  lx = (dx == 0) ? (hx - basis[0]) : basis[0];
	  for (dy=0; dy<2; dy++){
	    pby = pbc_box(b[1]+dy);
	    ly = (dy == 0) ? (hx - basis[1]) : basis[1];
	    for (dz=0; dz<2; dz++){
	      pbz = pbc_box(b[2]+dz);
	      lz = (dz == 0) ? (hx - basis[2]) : basis[2];
	      weight = lx*ly*lz/hx3;
	      w[id][pbx][pby][pbz] = weight;
	      rho[pbx][pby][pbz] +=  weight;
	    }
	  }
	}
}

// assign electron into bn*bn*bn boxes and store their idx
void update_box(double r[N][3], int box[bn][bn][bn][boxcap],int boxid[N][3],double rho[bn][bn][bn],double w[N][2][2][2]) {
	int i,j,k,realb[3],b[3],num;
	double basis[3];
	for (i=0; i<bn; i++){
		for (j=0; j<bn; j++){
			for (k=0; k<bn; k++){
				box[i][j][k][0]=0;
				rho[i][j][k] = 0.0;
			}
		}
	}
	printf("update box starts\n");
	for (i=0; i<N; i++) {
//		printf("i = %d \n",i);
		for (j=0; j<3; j++) {
			realb[j] = (int)floor(r[i][j]/hx);
			//b[j] = (int)floor(r[i][j]/hx) + grid_offset;
			b[j] = (int)floor(r[i][j]/hx);
//			printf("%d ",b[j]);
			boxid[i][j] = b[j];
//			printf("%d ",boxid[i][j]);
			basis[j] = r[i][j] - realb[j]*hx;
		}
//		printf("\n");
		num = box[b[0]][b[1]][b[2]][0]+1;
		box[b[0]][b[1]][b[2]][0] = num;
		box[b[0]][b[1]][b[2]][num] = i;
		charge_assign(b,basis,rho,i,w);	
	}
}

void setupKpoints(float kpoints[bn]){
	float k;
	int i;
	float nyquist = (float)bn/newR/2.0;
	for (i=0; i<(bn/2+1); i++){
		kpoints[i] = 2*PI*(float)i/(float)(bn/2)*nyquist;
		if (i!=(bn/2) && i!=0) {
			kpoints[bn-i] = kpoints[i];
		}
	}
}

__global__ void OutputToInput(hipfftComplex *f_fft_result,hipfftComplex *b_fft_in_x,hipfftComplex *b_fft_in_y,hipfftComplex *b_fft_in_z,float *d_kpoints) {
	int tx = blockIdx.x*blockDim.x + threadIdx.x;
	int ty = blockIdx.y*blockDim.y + threadIdx.y;
	int tz = blockIdx.z*blockDim.z + threadIdx.z;
	__shared__ float ks[bn];
	hipfftComplex in;
	float k2i,kx,ky,kz;
	int i,j,k;
	int b3 = bn/2+1;

	for (i=0; i<bn; i++){
		ks[i] = d_kpoints[i];
	}
	kx = ks[tx];
	ky = ks[ty];
	kz = ks[tz];
	int idx = 2*(tz + ty*b3 + tx*bn*b3);
	if(tx==0 && ty ==0 && tz==0) {
		k2i = 1.0;
	}
	else{
		k2i = 1.0/(kx*kx+ky*ky+kz*kz);
	}
	in.x = f_fft_result[idx].x;
	in.y = f_fft_result[idx].y;

	b_fft_in_x[idx].x = kx*k2i*in.y;
	b_fft_in_x[idx].y = -kx*k2i*in.x;
	b_fft_in_y[idx].x = ky*k2i*in.y;
	b_fft_in_y[idx].y = -ky*k2i*in.x;
	b_fft_in_z[idx].x = kz*k2i*in.y;
	b_fft_in_z[idx].y = -kz*k2i*in.x;	

}

void getGlobalField_cufft(double rho[bn][bn][bn],double field[3][bn][bn][bn]) {
	float h_input[bn][bn][bn],h_output[3][bn][bn][bn];
	float kpoints[bn];
	int i,j,k,idx;
	int b_complex = bn*bn*((int)bn/2+1);

	hipfftReal *d_input,*d_output_x,*d_output_y,*d_output_z,*d_kpoints;
	hipfftComplex 	*f_fft_result, *b_fft_in_x, *b_fft_in_y, *b_fft_in_z;
	hipfftHandle plan_forward, plan_backward_x, plan_backward_y, plan_backward_z;
	
	float elapsed = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	for(i=0; i<bn; i++){
		for(j=0; j<bn; j++){
			for(k=0; k<bn; k++){
				h_input[i][j][k] = rho[i][j][k];
			}
		}
	}

	printf("setting up K-points...\n");
	setupKpoints(kpoints);

	printf("cuFFT is starting...\n");
//	for(i=0; i<bn; i++){
//		printf("%11.3f ",h_input[0][0][i]);
//	}
//	printf("\n");

	//Allocate device momory
	hipMalloc((void**)&d_input, bn3*sizeof(hipfftReal));
	hipMalloc((void**)&d_output_x, bn3*sizeof(hipfftReal));
	hipMalloc((void**)&d_output_y, bn3*sizeof(hipfftReal));
	hipMalloc((void**)&d_output_z, bn3*sizeof(hipfftReal));
	hipMalloc((void**)&f_fft_result, b_complex*sizeof(hipfftComplex));
	hipMalloc((void**)&b_fft_in_x, b_complex*sizeof(hipfftComplex));
	hipMalloc((void**)&b_fft_in_y, b_complex*sizeof(hipfftComplex));
	hipMalloc((void**)&b_fft_in_z, b_complex*sizeof(hipfftComplex));
	hipMalloc((void**)&d_kpoints, bn*sizeof(hipfftReal));
	
	hipEventRecord(start,0);
	//Copy host momory to device
	hipMemcpy(d_input, &h_input[0][0][0], bn3, hipMemcpyHostToDevice);
	hipMemcpy(d_kpoints, &kpoints[0], bn, hipMemcpyHostToDevice);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);

	printf("The time for copy from host to device: %.3f ms\n", elapsed);

	//cuFFT plan
	hipfftPlan3d(&plan_forward, bn, bn, bn, HIPFFT_R2C);
	hipfftPlan3d(&plan_backward_x, bn, bn, bn, HIPFFT_C2R);
	hipfftPlan3d(&plan_backward_y, bn, bn, bn, HIPFFT_C2R);
	hipfftPlan3d(&plan_backward_z, bn, bn, bn, HIPFFT_C2R);

	printf("forward fft is starting...\n");
	hipEventRecord(start,0);
	//forward fft
	hipfftExecR2C(plan_forward, d_input, f_fft_result);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	printf("The time for Forward FFT: %.3f ms\n", elapsed);
	
	//transforming output_of_fft to input_of_ifft
	printf("output of fft ==> input of ifft\n");
	dim3 DimGrid(bn,1,1);
	dim3 DimBlock(1,bn,bn/2+1);
	OutputToInput<<<DimGrid,DimBlock>>>(f_fft_result,b_fft_in_x,b_fft_in_y,b_fft_in_z,d_kpoints);

	printf("backward fft is starting...\n");
	hipEventRecord(start,0);
	//ifft
	hipfftExecC2R(plan_backward_x, b_fft_in_x, d_output_x);
	hipfftExecC2R(plan_backward_y, b_fft_in_y, d_output_y);
	hipfftExecC2R(plan_backward_z, b_fft_in_z, d_output_z);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	printf("The time for Backward FFT: %.3f ms\n", elapsed);
	
	hipEventRecord(start,0);
	//Copy device memory to host
	hipMemcpy(&h_output[0][0][0][0], d_output_x, bn3, hipMemcpyDeviceToHost);
	hipMemcpy(&h_output[1][0][0][0], d_output_y, bn3, hipMemcpyDeviceToHost);
	hipMemcpy(&h_output[2][0][0][0], d_output_z, bn3, hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	printf("The time for copy data from device to Host: %.3f ms\n", elapsed);
//	for(i=0; i<bn; i++){
//		printf("%11.3f ",h_output[0][0][i]);
//	}
//	printf("\n");
	
	for(i=0; i<bn; i++){
		for(j=0; j<bn; j++){
			for(k=0; k<bn; k++){
				field[0][i][j][k] = (double)h_output[0][i][j][k];
				field[1][i][j][k] = (double)h_output[1][i][j][k];
				field[2][i][j][k] = (double)h_output[2][i][j][k];
			}
		}
	}

	//Destory cuFFT context
//	cufftDestroy(plan_forward);
//	cufftDestroy(plan_backward_x);
//	cufftDestroy(plan_backward_y);
//	cufftDestroy(plan_backward_z);
//
//	free(d_input);
//	free(d_output);
//	free(f_fft_result);
//	free(b_fft_in_x);
//	free(b_fft_in_y);
//	free(b_fft_in_z);

}

void getForce(double f[N][3],double r[N][3], int box[bn][bn][bn][boxcap],int boxid[N][3],double rho[bn][bn][bn],double w[N][2][2][2]) {
	int i,j,k,fx,b[3],dbx,dby,dbz,pbx,pby,pbz,nb_j;
	double rel[3], rel_c, rel_c2, fij[3], field[3][bn][bn][bn], ri[3], rj[3];
	double dummy;
	int jx,jy,jz,ijx,ijy,ijz;
	double jrel2;

//	getGlobalField(rho,field); 
//******[GPU] cufft******
	getGlobalField_cufft(rho,field);
//**********
	

//	#pragma omp parallel for private(bx,by,bz,pbx,pby,pbz,dbx,dby,dbz,nb_j,rel_c2,j,k,dummy)
	for (i=0; i<N; i++) {
		for (fx=0; fx<3; fx++){
			b[fx] = boxid[i][fx];
			ri[fx] = r[i][fx];
		}
		for (dbx=0; dbx<3; dbx++){
			pbx = pbc_box(b[0]+dbx-1);
			for (dby=0; dby<3; dby++){
				pby = pbc_box(b[1]+dby-1);
				for (dbz=0; dbz<3; dbz++){
					pbz = pbc_box(b[2]+dbz-1);
					for (j=1; j<(box[pbx][pby][pbz][0]+1); j++){
						nb_j = box[pbx][pby][pbz][j];
						if (nb_j != i) {
//*******PP*******
							rel_c2 = 0.0;
							for (k=0; k<3; k++) {
								rj[k] = r[nb_j][k];
								dummy = ri[k]-rj[k];
								rel[k] = dummy - floor(dummy/newR)*newR;
								rel_c2 += rel[k]*rel[k];
							}
							for(k=0; k<3; k++) {
							//	if (rel_c2 < 1e-6) {rel_c2 = 1e-6; printf("small number!");}
								f[i][k] += rel[k]/pow(rel_c2,1.5);
							}
//*******PM*******
							//cancel nb_j on neighboring grid point for PM
							for (jx=0; jx<2; jx++){
								for (jy=0; jy<2; jy++){
									for (jz=0; jz<2; jz++){
										//get to each neighboring gird point
										ijx = pbc_box(b[0]+jx);
										ijy = pbc_box(b[1]+jy);
										ijz = pbc_box(b[2]+jz);
										jrel2 = pow((rj[0]-ijx*hx),2.0)+pow((rj[1]-ijy*hx),2.0)+pow((rj[2]-ijz*hx),2.0);
										field[0][ijx][ijy][ijz] -= (ijx*hx-rj[0])/pow(jrel2,1.5);
										field[1][ijx][ijy][ijz] -= (ijy*hx-rj[1])/pow(jrel2,1.5);	
										field[2][ijx][ijy][ijz] -= (ijz*hx-rj[2])/pow(jrel2,1.5);	
									}
								}
							}
						}
					}
				}
			}
		}
		//PM
		for (dbx=0; dbx<2; dbx++){
			pbx = pbc_box(b[0]+dbx);
			for (dby=0; dby<2; dby++){
				pby = pbc_box(b[1]+dby);
				for (dbz=0; dbz<2; dbz++){
					pbz = pbc_box(b[2]+dbz);
					for (fx=0; fx<3; fx++){
						f[i][fx] += w[i][dbx][dby][dbz]*field[fx][dbx][dby][dbz];
					}
				}
			}
		}

	}
	printf("getforce finish\n");

}

void verlet(double r[N][3],double v[N][3],double f[N][3], int box[bn][bn][bn][boxcap],int boxid[N][3],double rho[bn][bn][bn],double w[N][2][2][2], double dt){
	int i,j;
	double hdtm = 0.5*dt/m;
	for (i=0; i<N; i++) {
		for (j=0; j<3; j++) {
			v[i][j] += f[i][j]*hdtm;
			r[i][j] += v[i][j]*dt;
			r[i][j] = r[i][j]-floor(r[i][j]/newR)*newR;
			f[i][j] = 0.0; //setup for force calculation
		}
	}
	update_box(r,box,boxid,rho,w);
	getForce(f,r,box,boxid,rho,w);
	for (i=0; i<N; i++) {
		for (j=0; j<3; j++) {
			v[i][j] += f[i][j]*hdtm;
		}
	}
	printf("update v finish\n");
}

void inputbinning(double r[N][3],double rtest[N][3]){
  //STEP 1: ALLOCATE
  double *r_d;
  int N3size = sizeof(double)*N*3;
  hipMalloc((void **) &r_d, N3size);
  
  //STEP 2: TRANSFER
  hipMemcpy(r_d, r, N3size, hipMemcpyHostToDevice);

  //STEP 3: SET UP
  dim3 blockSize(N,1,1);
  dim3 gridSize(1,3,1);

  //STEP 4: RUN
  theKernel<<<gridSize, blockSize>>>(r_d);
  
  //STEP 5: TRANSFER
  printGrid(r);
  hipMemcpy(rtest, r_d, N3size, hipMemcpyDeviceToHost);
  printf("--------------------\n");
  printGrid(r);
  printf("--------------------\n");
  printGrid(rtest);
}

int main() {
	double R[N][3] = {{0.0}}, V[N][3] = {{0.0}}, F[N][3]={{0.0}};
	int box[bn][bn][bn][boxcap]; // need to go to class or dynamic array later 
	// [0] is for number of electrons in the box, and then [1]-[number] is the electron id
	int boxid[N][3];
	int i, iter;
	int numb, check;
	double dt = 2e-4, realt = 0.0;
	double r0,r1,r2,rel0,rel1,rel2;
	double rho[bn][bn][bn], w[N][2][2][2];

	// Initialization in a sphere
	srand(time(NULL));
	numb = 0;
	while (numb < N) {
		r0 = newR*getrand();
		r1 = newR*getrand();
		r2 = newR*getrand();

		check = 0;
		if (sqrt(r0*r0+r1*r1+r2*r2) < newR) {
			for (i = 0; i < numb; i++){
				rel0 = R[i][0] - r0;
				rel1 = R[i][1] - r1;
				rel2 = R[i][2] - r2;
				if (sqrt(rel0*rel0+rel1*rel1+rel2*rel2) < cutoff) {
					check = 1;
					break;
				}				
			}
			if (check == 0) {
				R[numb][0] = r0;
				R[numb][1] = r1;
				R[numb][2] = r2;
				numb += 1;
			}
		}
	}
	
	for (iter = 0; iter< Ntime; iter++) {
		printf("iter %d: \n", iter);
		verlet(R,V,F,box,boxid,rho,w,dt);
		realt += dt;
		printf("update realtime \n");
	}

//******[GPU] Input binning for local correct ******

//	double rtest[N][3]={{0.0}};
//	inputbinning(R,rtest);

//**********


}
